
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define N 512

__global__ void deviceAdd(int* a, int* b, int* c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void hostAdd(int* a, int* b, int* c){
	for(int index = 0; index < N; index++){
		c[index] = a[index] + b[index];
	}
}

void populateArray(int* data){
	for(int index = 0; index < N; index++){
		data[index] = index;
	}
}

void print(int* a, int* b, int* c){
	for(int index = 0; index < N; index++){
		printf("\n %d + %d = %d", a[index], b[index], c[index]);
	}
}

int main(int argc, char** argv){
	int *a, *b, *c;
	int *devA, *devB, *devC;
	int size = N * sizeof(int);

	// Allocate space for host copies of a, b, c
	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	populateArray(a);
	populateArray(b);

	// Allocate space for device copies of a, b, c
	// cudaMalloc ( void** devPtr, size_t size );
	hipMalloc((void**)&devA, size);
	hipMalloc((void**)&devB, size);
	hipMalloc((void**)&devC, size);

	// cudaMemcpy(void* dst, const void* src, size_t count, cudaMemcpyKind kind);
	hipMemcpy(devA, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(devB, b, N * sizeof(int), hipMemcpyHostToDevice);
	
	// N blocks - 1 Thread per block
	deviceAdd<<<N, 1>>>(devA, devB, devC);
	
	// Copy results to host
	hipMemcpy(c, devC, N * sizeof(int), hipMemcpyDeviceToHost);

	
	print(a, b, c);
	
	free(a);
	free(b);
	free(c);

	// Free device memory
	hipFree(devA);
	hipFree(devB);
	hipFree(devC);

	return 0;
}
